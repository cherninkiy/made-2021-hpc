#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc.hpp>

using namespace cv;
using namespace std;

const int NUM_THREADS = 16;
const int KERNEL_SIZE = 5;

__global__ void median2d(int rows, int cols, int channels, const float *image, float *buffer) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int c = threadIdx.z;
    int b = KERNEL_SIZE / 2;
    const int n = KERNEL_SIZE * KERNEL_SIZE;

    double pix, tmp;
    int counter = 0;
    double arr[n] = {0};
    if (row > b && row < (rows - b) && col > b && col < (cols - b))
    {
        for (int k = -b; k <= b; k++)
        {
            for (int l = -b; l <= b; l++)
            {
                arr[counter] = image[(row + l) * cols * channels + (col + k) * channels + c];
                counter++;
            }
        }
        // sort
        for (int i = 0; i < n; i++)
        {
            for (int j = i + 1; j < n; j++)
            {
                if (arr[i] > arr[j])
                {
                    tmp = arr[i];
                    arr[i] = arr[j];
                    arr[j] = tmp;
                }
            }
        }
        if (n % 2)
            pix = arr[b];
        else
            pix = (arr[b] + arr[b+1])/2;
    }
    else
    {
        pix = image[row * cols * channels + col * channels + c];
    }
    buffer[row * cols * channels + col * channels + c] = pix;
}

hipError_t applyMedian2d(int rows, int cols, int channels, const uchar* image, float* buffer)
{
    int num_pixels = rows * cols * channels;
    int num_bytes = num_pixels * sizeof(float);

    hipError_t error;

    float* img_array;
    error = hipMalloc(&img_array, num_bytes);
    if (error != hipError_t::hipSuccess)
    {
        return error;
    }

    float* buf_array;
    error = hipMalloc(&buf_array, num_bytes);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        return error;
    }

    error = hipMemcpy(img_array, image, num_bytes, hipMemcpyHostToDevice);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        return error;
    }

    int grid_width = (cols + NUM_THREADS - 1) / NUM_THREADS;
    int grid_height = (rows + NUM_THREADS - 1) / NUM_THREADS;
    const dim3 grid_size(grid_width, grid_height);
    const dim3 block_size(NUM_THREADS, NUM_THREADS, channels);

    median2d<<<grid_size, block_size>>>(rows, cols, channels, img_array, buf_array);

    error = hipPeekAtLastError();
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        return error;
    }

    error = hipMemcpy(buffer, buf_array, num_bytes, hipMemcpyDeviceToHost);

    hipFree(img_array);
    hipFree(buf_array);
    return error;
}

int main(int argc, char* argv[])
{
    Mat image = imread("images/lena.jpg");
    image.convertTo(image, CV_32F, 1.0 / 255);

    vector<float> buffer(image.rows * image.cols * image.channels(), 0.0f);

    auto error = applyMedian2d(image.rows, image.cols, image.channels(), &image.data[0], buffer.data());
    if (error != hipError_t::hipSuccess)
    {
        cout << hipGetErrorString(error) << endl;
        return 1;
    }

    Mat result(image.rows, image.cols, CV_32FC3, buffer.data());
    result.convertTo(result, CV_8U, 255);

    if (!imwrite("images/lena_median.jpg", result))
    {
        cout << "Cannot save image" << endl;
    }

    cout << "Output: images/lena_median.jpg" << endl;

    return 0;
}
