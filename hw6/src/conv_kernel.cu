#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc.hpp>

using namespace cv;
using namespace std;

const int NUM_THREADS = 16;


__global__ void conv2d(int rows, int cols, int channels, const float *image,
    int kernel_size, const float* kernel, float norm, float *buffer)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
	int c = threadIdx.z;
    int b = kernel_size / 2;

    double pix = 0;
    if (row > b && row < (rows - b) && col > b && col < (cols - b))
    {
        for (int k = -b; k <= b; k++)
        {
            for (int l = -b; l <= b; l++)
            {
                pix += 1/norm * kernel[((k + 1) * kernel_size + l) % kernel_size] 
                        * image[(row + l) * cols * channels + (col + k) * channels + c];
            }
        }
    }
    else
    {
        pix = image[row * cols * channels + col * channels + c];
    }
    pix = fminf(fmaxf(pix, 0.0f), 1.0f);
    buffer[row * cols * channels + col * channels + c] = pix;
}

hipError_t applyConv2d(int rows, int cols, int channels, const uchar* image,
    int kernel_size, const float* kernel, float norm, float* buffer)
{
    int num_pixels = rows * cols * channels;
    int num_bytes = num_pixels * sizeof(float);
    int kernel_bytes = kernel_size * sizeof(float);

    hipError_t error;

    float* img_array;
    error = hipMalloc(&img_array, num_bytes);
    if (error != hipError_t::hipSuccess)
    {
        return error;
    }

    float* buf_array;
    error = hipMalloc(&buf_array, num_bytes);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        return error;
    }

    float* kernel_array;
    error = hipMalloc(&kernel_array, kernel_bytes);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        return error;
    }

    error = hipMemcpy(img_array, image, num_bytes, hipMemcpyHostToDevice);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        hipFree(kernel_array);
        return error;
    }

    error = hipMemcpy(kernel_array, kernel, kernel_bytes, hipMemcpyHostToDevice);
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        hipFree(kernel_array);
        return error;
    }

    int grid_width = (cols + NUM_THREADS - 1) / NUM_THREADS;
    int grid_height = (rows + NUM_THREADS - 1) / NUM_THREADS;
    const dim3 grid_size(grid_width, grid_height);
    const dim3 block_size(NUM_THREADS, NUM_THREADS, channels);

    conv2d<<<grid_size, block_size>>>(rows, cols, channels, img_array,
            kernel_size, kernel_array, norm, buf_array);

    error = hipPeekAtLastError();
    if (error != hipError_t::hipSuccess)
    {
        hipFree(img_array);
        hipFree(buf_array);
        hipFree(kernel_array);
        return error;
    }

    error = hipMemcpy(buffer, buf_array, num_bytes, hipMemcpyDeviceToHost);

    hipFree(img_array);
    hipFree(buf_array);
    hipFree(kernel_array);
    return error;
}

int main(int argc, char* argv[])
{
    Mat image = imread("images/lena.jpg");
    image.convertTo(image, CV_32F, 1.0 / 255);

    // Blur 5x5
    {
        vector<float> kernel = {
                1.0, 1.0, 1.0, 1.0, 1.0,
                1.0, 1.0, 1.0, 1.0, 1.0,
                1.0, 1.0, 1.0, 1.0, 1.0,
                1.0, 1.0, 1.0, 1.0, 1.0,
                1.0, 1.0, 1.0, 1.0, 1.0
            };

        float norm = kernel.size() * kernel.size();

        vector<float> buffer(image.rows * image.cols * image.channels(), 0.0f);

        auto error = applyConv2d(image.rows, image.cols, image.channels(), &image.data[0],
                            kernel.size(), kernel.data(), norm, buffer.data());
        if (error != hipError_t::hipSuccess)
        {
            cout << hipGetErrorString(error) << endl;
            return 1;
        }

        Mat result(image.rows, image.cols, CV_32FC3, buffer.data());
        result.convertTo(result, CV_8U, 255);

        if (!imwrite("images/lena_blur.jpg", result))
        {
            cout << "Cannot save image" << endl;
        }

        cout << "Output: images/lena_blur.jpg" << endl;
    }

    // Conv 3x3
    {
        vector<float> kernel = {
                0.5f, 0.5f, 0.5f,
                1.0f, 1.0f, 1.0f,
                0.5f, 0.5f, 0.5f
            };

        float norm = 0.0;
        for (int i = 0; i < kernel.size(); ++i) {
            norm += kernel[i];
        }

        vector<float> buffer(image.rows * image.cols * image.channels(), 0.0f);

        auto error = applyConv2d(image.rows, image.cols, image.channels(), &image.data[0],
                            kernel.size(), kernel.data(), norm, buffer.data());
        if (error != hipError_t::hipSuccess)
        {
            cout << hipGetErrorString(error) << endl;
            return 1;
        }

        Mat result(image.rows, image.cols, CV_32FC3, buffer.data());
        result.convertTo(result, CV_8U, 255);

        if (!imwrite("images/lena_conv.jpg", result))
        {
            cout << "Cannot save image" << endl;
        }

        cout << "Output: images/lena_conv.jpg" << endl;
    }

    return 0;
}
